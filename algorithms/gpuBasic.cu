#include "hip/hip_runtime.h"
#include "../gameLogic/GameInstance.h"
#include "../tools/Timer.h"
#include <hip/hip_runtime.h>

__global__ void gameKernel(bool* gameArea, bool* nextIteration, int columns, int rows) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //sprawdzenie, czy wątek nie wyszedł za plansze
    if (x >= columns || y >= rows) return;

    int neighbors = 0;
    for (int dy=-1; dy<=1; dy++) {
        for (int dx=-1; dx<=1; dx++) {
            if(dx==0 && dy==0) continue;
            int nx = x + dx;
            int ny = y + dy;
            if(nx>=0 && nx<columns && ny>=0 && ny<rows) //sprawdzenie, czy sąsiad mieści się na planszy
                neighbors += gameArea[ny*columns + nx];
        }
    }

    nextIteration[y*columns + x] = (neighbors == 3 || (neighbors == 2 && gameArea[y*columns + x]));
}

void gpuBasic(GameInstance& game) {
    int columns = game.getColumns();
    int rows = game.getRows();

    // Alokacja pamięci GPU
    bool* d_gameArea;
    bool* d_nextIteration;
    size_t size = columns * rows * sizeof(bool);
    hipMalloc(&d_gameArea, size);
    hipMalloc(&d_nextIteration, size);

    // kopiowanie danych z 2D na 1D
    bool* flat = new bool[columns*rows];
    for(int y=0;y<rows;y++)
        for(int x=0;x<columns;x++)
            flat[y*columns + x] = game.getGameArea()[y][x];

    // kopiowanie planszy z CPU na GPU
    hipMemcpy(d_gameArea, flat, size, hipMemcpyHostToDevice);

    dim3 blockSize(16,16);
    dim3 gridSize((columns+15)/16, (rows+15)/16);
    gameKernel<<<gridSize, blockSize>>>(d_gameArea, d_nextIteration, columns, rows);
    hipDeviceSynchronize(); //czekanie aż wszystkie wątki zakończą działanie kernela

    // Zapis wyników z powrotem do GameInstance
    hipMemcpy(flat, d_nextIteration, size, hipMemcpyDeviceToHost);
    for(int y=0;y<rows;y++)
        for(int x=0;x<columns;x++)
            game.getGameArea()[y][x] = flat[y*columns + x];

    delete[] flat;
    hipFree(d_gameArea);
    hipFree(d_nextIteration);
}

double gpuBasic(GameInstance& game, int iterations) {
    int columns = game.getColumns();
    int rows = game.getRows();

    // Alokacja pamięci GPU
    bool* d_gameArea;
    bool* d_nextIteration;
    size_t size = columns * rows * sizeof(bool);
    hipMalloc(&d_gameArea, size);
    hipMalloc(&d_nextIteration, size);

    // kopiowanie danych z 2D na 1D
    bool* flat = new bool[columns * rows];
    for(int y=0; y<rows; y++)
        for(int x=0; x<columns; x++)
            flat[y*columns + x] = game.getGameArea()[y][x];

    hipMemcpy(d_gameArea, flat, size, hipMemcpyHostToDevice);

    dim3 blockSize(16,16);
    dim3 gridSize((columns+15)/16, (rows+15)/16);

    Timer timer;
    timer.Start();

    //uruchomienie kernela dla kazdej iteracji
    for(int i=0; i<iterations; i++) {
        gameKernel<<<gridSize, blockSize>>>(d_gameArea, d_nextIteration, columns, rows);
        hipDeviceSynchronize();

        // Zamiana wskaźników zamiast kopiowania w każdej iteracji
        std::swap(d_gameArea, d_nextIteration);
    }

    hipMemcpy(flat, d_gameArea, size, hipMemcpyDeviceToHost);

    // Zapis wyników z powrotem do GameInstance
    for(int y=0; y<rows; y++)
        for(int x=0; x<columns; x++)
            game.getGameArea()[y][x] = flat[y*columns + x];

    const double output = timer.Stop();

    delete[] flat;
    hipFree(d_gameArea);
    hipFree(d_nextIteration);

    return output;
}
